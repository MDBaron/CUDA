#include "hip/hip_runtime.h"
/*
 * 
 * Matthew Baron
 * Homework #4
 * 3/16/2015
 * CSCI 4150
 * CUDA Version #1
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <ctime>
#include <iomanip>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 16

using namespace std;

//Row-Major Matrix struct 
typedef struct {
	int width;
	int height;
	int* elements;
} Matrix;

__global__ void MatrixMultiplyKernel(Matrix A, Matrix B, Matrix C);

 
void MatrixMultiply(Matrix matA, Matrix matB, Matrix matC){

  Matrix d_A, d_B, d_C;

  int value = 16384;//Multiple of 16
  d_A.height = value;
  d_A.width = value;
  
  std::cout << "Matrix d_A Values Set" << std::endl;  
  
  d_B.height = value;
  d_B.width = value;
  
  std::cout << "Matrix d_B Values Set" << std::endl;

  d_C.height = value;
  d_C.width = value;
  
  std::cout << "Matrix d_C Values Set" << std::endl;

  /* Allocate and copy memory to DEVICE */
  size_t size = value * value * sizeof(int);
  hipMalloc(&d_A.elements, size);
  hipMemcpy(d_A.elements, matA.elements, size, hipMemcpyHostToDevice);
  hipMalloc(&d_B.elements, size);
  hipMemcpy(d_B.elements, matB.elements, size, hipMemcpyHostToDevice);
  hipMalloc(&d_C.elements, size);

  std::cout << "Memory Allocation on DEVICE Complete" << std::endl;
  
  /* Get dem thread blocks allocated */
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((value + dimBlock.x -1) / dimBlock.x, (value + dimBlock.y - 1) / dimBlock.y);

  std::cout << "Begin Call to Kernel for Matrix Multiplication.... " << std::endl;
  MatrixMultiplyKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
  hipDeviceSynchronize();
  
  std::cout << "Matrix Multiplication Complete" << std::endl;
  std::cout << "Copying Device Matrix d_C back to Host C..." << std::endl;  

   /* Copy results from DEVICE to HOST */
  hipMemcpy(matC.elements, d_C.elements, size, hipMemcpyDeviceToHost);
  
  std::cout << "Matrix Copy Complete" << std::endl;

  /*  Deallocate Memory on DEVICE */
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  
  std::cout << "Absolving Memory Allocations..." << std::endl;

}//MatrixMultiplyKernel 
 
__global__ void MatrixMultiplyKernel(Matrix A, Matrix B, Matrix C){
	int sumValue = 0;
	int col = blockIdx.x * blockDim.x + threadIdx.x;//Thread association for columns
	int row = blockIdx.y * blockDim.y + threadIdx.y;//Thread association for rows
	if(col > B.width || row > A.height){
		return;
	}//Bounds Checking
	for(int i = 0; i < A.width; i++){
		sumValue += A.elements[(row * A.width) + i] * B.elements[(i * B.width) + col];
	}
	C.elements[(row * C.width) + col] = sumValue;//Store summation in new matrix
}//global Kernel


int main(void){

  int value = 16384;//Multiple of 16
  
  Matrix matA, matB, matC;
  matA.height = value;
  matA.width = value;
  matA.elements = (int*)malloc(matA.width * matA.height * sizeof(int)); //String of elements that represent Row-Major Matrix
  
  std::cout << "Matrix A Allocations Complete" << std::endl;  

  matB.height = value;
  matB.width = value;
  matB.elements = (int*)malloc(matB.width * matB.height * sizeof(int)); //String of elements that represent Row-Major Matrix
  
  std::cout << "Matrix B Allocations Complete" << std::endl;  

  matC.height = value;
  matC.width = value;
  matC.elements = (int*)malloc(matC.width * matC.height * sizeof(int)); //String of elements that represent Row-Major Matrix
  
  std::cout << "Matrix C Allocations Complete" << std::endl;
  std::cout << "Current Value: " << value << std::endl;
   
  //Fill matrices with random data
  srand(time(NULL));
  
  for(int p = 0; p < value; ++p){
    for(int q = 0; q < value; ++q){
      matA.elements[(p * value) + q] = rand();
      matB.elements[(p * value) + q] = rand();
      //std::cout << p << " " << q << std::endl;
    }//Q
  }//P

  std::cout << "Random Data Fill for Matrices Complete " << std::endl;

  //Declare Time Events
  hipEvent_t start, stop; 
  float time; 
  
  //Create CUDA Time Events
  hipEventCreate(&start); 
  hipEventCreate(&stop); 
  
  std::cout << "Begin Matrix Multiply:.... " << std::endl;

  //Begin Recording
  hipEventRecord( start, 0 );   

  //Perform Kernel Operations
  MatrixMultiply(matA, matB, matC);

  //Halt Time Event Recording
  hipEventRecord( stop, 0 ); 
  hipEventSynchronize( stop ); 
  
  std::cout << "Job Complete: " << std::endl;  

  //Calculate and Store Time in CUDA Elapsed Time
  hipEventElapsedTime( &time, start, stop ); 
  
  //Free Event Memory
  hipEventDestroy( start ); 
  hipEventDestroy( stop );

  printf("Elapsed Time : %.*e ms/n"  , time);
  std::cout << std::endl;

  /* 

  C Program Average time elapsed is ~30secs on 1000 x 1000 matrix
  CUDA Program is returning 0.0000000000000000000e+00 ms on maximum decimal (Unknown Reasons)
  * Execution for 10,000+ is easily less than 2secs for CUDA which would mean substantial performance increase  

*/

  return 0;

}//main
